#include "hip/hip_runtime.h"
#include "bit_helper.cu"
#include "quant_kernel.h"
#include "sim_helper.cu"

// quantize a float into a floating point with [exp_bits] exponent and
// [man_bits] mantissa
__global__ void block_kernel_stochastic(float *__restrict__ a,
                                        int *__restrict__ r, float *o, int size,
                                        float *__restrict__ max_entry,
                                        int man_bits) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    unsigned int max_entry_bits = FLOAT_TO_BITS(&max_entry[index]);
    unsigned int max_exp = max_entry_bits << 1 >> 24 << 23;
    float base_float = 6 * BITS_TO_FLOAT(&max_exp);

    float target_rebase = a[index] + base_float;
    unsigned int target_bits = FLOAT_TO_BITS(&target_rebase);
    unsigned int rand_prob = (unsigned int)r[index];
    unsigned int quantized =
        round_bitwise_stochastic(target_bits, rand_prob, man_bits);
    float quantize_float = BITS_TO_FLOAT(&quantized) - base_float;

    unsigned int quantize_bits = FLOAT_TO_BITS(&quantize_float);
    unsigned int clip_quantize =
        clip_max_exponent(man_bits - 2, max_exp, quantize_bits);
    quantize_float = BITS_TO_FLOAT(&clip_quantize);
    o[index] = quantize_float;
  }
}

// quantize a float into a floating point with [exp_bits] exponent and
// [man_bits] mantissa
__global__ void block_kernel_nearest(float *__restrict__ a, float *o, int size,
                                     float *__restrict__ max_entry,
                                     int man_bits) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    unsigned int max_entry_bits = FLOAT_TO_BITS(&max_entry[index]);
    unsigned int max_exp = max_entry_bits << 1 >> 24 << 23;
    float base_float = 6 * BITS_TO_FLOAT(&max_exp);

    float target_rebase = a[index] + base_float;
    unsigned int target_bits = FLOAT_TO_BITS(&target_rebase);
    unsigned int quantized = round_bitwise_nearest(target_bits, man_bits);
    float quantize_float = BITS_TO_FLOAT(&quantized) - base_float;

    unsigned int quantize_bits = FLOAT_TO_BITS(&quantize_float);
    unsigned int clip_quantize = clip_max_exponent(
        man_bits - 2, max_exp, quantize_bits); // sign bit, virtual bit
    quantize_float = BITS_TO_FLOAT(&clip_quantize);

    o[index] = quantize_float;
  }
}

// quantize a float into a floating point with [exp_bits] exponent and
// [man_bits] mantissa
__global__ void block_kernel_sim_stochastic(float *__restrict__ a,
                                            float *__restrict__ r, float *o,
                                            int size, float *max_entry,
                                            int wl) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    int exponent = ((int)extract_exponent(max_entry));
    int sigma = exponent - (wl - 1);
    o[index] = round(a[index], r[index], sigma);
  }
}

// quantize a float into a floating point with [exp_bits] exponent and
// [man_bits] mantissa
__global__ void block_kernel_sim_nearest(float *__restrict__ a, float *o,
                                         int size, float *max_entry, int wl) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    int exponent = ((int)extract_exponent(max_entry));
    int sigma = exponent - (wl - 1);
    o[index] = nearest_round(a[index], sigma);
  }
}
